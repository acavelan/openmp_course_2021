#include "hip/hip_runtime.h"
#include <cstdio>
#include "utils.hpp"

__global__ void cuda_mul(float* m1, float* m2, float* m3, int n) {

    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    for (int i = 0; i < n; i++)
        m3[row * n + col] += m1[row * n + i] * m2[i * n + col];
}

void mul(const float *m1, const float *m2, float *m3, int n)
{
	float *dm1 = nullptr, *dm2 = nullptr, *dm3 = nullptr;

	hipMalloc((void**)&dm1, sizeof(float) * n * n);
	hipMalloc((void**)&dm2, sizeof(float) * n * n);
	hipMalloc((void**)&dm3, sizeof(float) * n * n);

	hipMemcpy(dm1, m1, sizeof(float) * n * n, hipMemcpyHostToDevice);
	hipMemcpy(dm2, m2, sizeof(float) * n * n, hipMemcpyHostToDevice);
	hipMemcpy(dm3, m3, sizeof(float) * n * n, hipMemcpyHostToDevice);

	dim3 blockSize = dim3(16, 16);
	dim3 gridSize = dim3(n / blockSize.x, n/ blockSize.y);
	
	cuda_mul<<<gridSize, blockSize>>>(dm1, dm2, dm3, n);

	hipMemcpy(m3, dm3, sizeof(float) * n * n, hipMemcpyDeviceToHost);

	hipFree(dm1);
	hipFree(dm2);
	hipFree(dm3);
}

int main(int argc, char **argv)
{
	int n = 64;
	if(argc > 1)
		n = atoi(argv[1]);

	srand(12);

	float *m1 = nullptr, *m2 = nullptr, *m3 = nullptr;

	hipError_t err = hipHostMalloc((void**)&m1, sizeof(float) * n * n);
	if(err) printf("Error status is %s\n", hipGetErrorString(err));
	err = hipHostMalloc((void**)&m2, sizeof(float) * n * n);
	if(err) printf("Error status is %s\n", hipGetErrorString(err));
	err = hipHostMalloc((void**)&m3, sizeof(float) * n * n);
	if(err) printf("Error status is %s\n", hipGetErrorString(err));

	random(m1, n);
	random(m2, n);
	random(m3, n);

	zeros(m3, n);

	TimePoint tstart, tstop;

	tstart = Clock::now();
	mul(m1, m2, m3, n);
	tstop = Clock::now();

	//printf("checksum: %f -- Time: %fms\n", checksum(m3, n), elapsedTime(tstart, tstop));
	printf("%f %f\n", checksum(m3, n), elapsedTime(tstart, tstop));

	hipHostFree(m1);
	hipHostFree(m2);
	hipHostFree(m3);
}
